#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void erode ( unsigned char * data,   unsigned char * end_result, std::size_t cols, std::size_t rows) {

  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if ( i > 1 && i < (cols - 2) && j > 1 && j < (rows - 2)) {
    for (auto c = 0; c < 3; ++c){
      auto hu = umax(data[((j - 1) * cols + i - 1) * 3 + c], data[((j - 1) * cols + i + 1) * 3 + c]);
      hu = umax(hu, data[(( j - 1) * cols + i) * 3 + c]);
      auto mu = umax(data[( j * cols + i - 1) * 3 + c],  data[( j * cols + i + 1) * 3 + c]);
      mu = umax(mu, data[( j * cols + i) * 3 + c]);
      auto bu = umax(data[((j + 1) * cols + i - 1) * 3 + c], data[((j + 1) * cols + i + 1) * 3 + c]);
      bu = umax(bu, data[((j + 1) * cols + i) * 3 + c]);
      auto res = umax (hu, mu);
      res = umax (res, bu);
      end_result[(j * cols + i) * 3 + c] = res;
    }
  }

}

int main()
{
  cv::Mat img_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = img_in.data;
  auto rows = img_in.rows;
  auto cols = img_in.cols;
  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat img_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * end_result;
  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &end_result, 3 * rows * cols );
  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 dim1( 32, 32 );
  dim3 dim2(( cols - 1) / dim1.x + 1 , ( rows - 1 ) / dim1.y + 1 );
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );
  erode<<< dim2, dim1 >>>( rgb_d, end_result, cols, rows );
  hipMemcpy(g.data(), end_result, 3 * rows * cols, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  auto hipError_t = hipGetLastError();
  if (hipError_t != hipSuccess){
    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }
  else {
    std::cout << "No Errors!" << std::endl;
  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );
  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "Total: " << duration << "ms\n";
  cv::imwrite( "ErosionOutput.jpg", img_out );
  hipFree( rgb_d);
  hipFree ( end_result);
  return 0;
}
