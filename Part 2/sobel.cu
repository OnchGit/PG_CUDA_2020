#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void sobel ( unsigned char * data,   unsigned char * end_result, std::size_t cols, std::size_t rows) {

  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if ( i > 0 && i < (cols - 1) && j > 0 && j < (rows - 1)) {
		for (auto c = 0; c < 3; ++c){
		  auto op1 = data[((j - 1) * cols + i - 1) * 3 + c] - data[((j - 1) * cols + i + 1) * 3 + c]
		  + 2 * data[( j * cols + i - 1) * 3 + c] - 2 * data[( j * cols + i + 1) * 3 + c]
		  + data[((j + 1) * cols + i - 1) * 3 + c] - data[((j + 1) * cols + i + 1) * 3 + c];
		  auto op2 = data[((j - 1) * cols + i - 1) * 3 + c] - data[((j + 1) * cols + i - 1) * 3 + c]
		  + 2 * data[((j - 1) * cols + i    ) * 3 + c] - 2 * data[((j + 1) * cols + i ) * 3 + c]
		  + data[((j - 1) * cols + i + 1) * 3 + c] - data[((j + 1) * cols + i + 1) * 3 + c];
		  auto res = op1*op1 + op2*op2;
		  res = res > 255*255 ? res = 255*255 : res;
		  end_result[(j * cols + i) * 3 + c] = sqrt((float) res);
		}
	}
}

int main()
{
  cv::Mat img_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = img_in.data;
  auto rows = img_in.rows;
  auto cols = img_in.cols;
  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat img_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * end_result;
  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &end_result, 3 * rows * cols );
  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 dim2( 32, 32 );
  dim3 dim1(( cols - 1) / dim2.x + 1 , ( rows - 1 ) / dim2.y + 1 );
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );
  sobel<<< dim1, dim2 >>>( rgb_d, end_result, cols, rows );
  hipMemcpy(g.data(), end_result, 3 * rows * cols, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  auto hipError_t = hipGetLastError();
  if (hipError_t != hipSuccess){
    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }
  else {
    std::cout << "No Errors!" << std::endl;
  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );
  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "Total: " << duration << "ms\n";
  cv::imwrite( "SobelOutput.jpg", img_out );
  hipFree( rgb_d);
  hipFree ( end_result);
  return 0;
}
