#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
      307 * rgb[ 3 * ( j * cols + i ) ]
      + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
      + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
    ) / 1024;
  }
}

int main()
{
  cv::Mat img_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = img_in.data;
  auto rows = img_in.rows;
  auto cols = img_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat img_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * g_d;
  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &g_d, rows * cols );
  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 dim1( 32, 32 );
  dim3 dim2(( cols - 1) / dim1.x + 1 , ( rows - 1 ) / dim1.y + 1 );
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );
  grayscale<<< dim2, dim1 >>>( rgb_d, g_d, cols, rows );
  hipMemcpy(g.data(), g_d, rows * cols, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  auto hipError_t = hipGetLastError();
  if (hipError_t != hipSuccess){
    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }
  else {
    std::cout << "No Errors!" << std::endl;
  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );
  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "Total: " << duration << "ms\n";
  cv::imwrite( "GSOutput.jpg", img_out );
  hipFree( rgb_d);
  hipFree( g_d);
  return 0;
}
