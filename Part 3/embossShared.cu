#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void embossShared ( unsigned char * data,   unsigned char * out, std::size_t w, std::size_t h) {

  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  auto op1 = threadIdx.x;
  auto op2 = threadIdx.y;

  extern __shared__ unsigned char sharedExt[];

  if( i < w && j < h ) {
    sharedExt[3 * (op2 * blockDim.x + op1) ] = data[ 3 * ( j * w + i ) ];
    sharedExt[3 * (op2 * blockDim.x + op1) + 1 ] = data[ 3 * ( j * w + i ) + 1];
    sharedExt[3 * (op2 * blockDim.x + op1) + 2 ] = data[ 3 * ( j * w + i ) + 2 ];
    __syncthreads();
    auto op3 = blockDim.x;
    if( op1 > 0 && op1 < (blockDim.x - 1) && op2 > 0 && op2 < (blockDim.y - 1) ){
      for (auto c = 0; c < 3; ++c){
          auto op4 = sharedExt[((op2 - 1) * op3 + op1 - 1) * 3 + c] * -18 + sharedExt[((op2 - 1) * op3 + op1 + 1) * 3 + c] * 0
          + sharedExt[( op2 * op3 + op1 - 1) * 3 + c] * -9 + sharedExt[( op2 * op3 + op1 + 1) * 3 + c] * 9
		  + sharedExt[((op2 + 1) * op3 + op1 - 1) * 3 + c] * 0  + sharedExt[((op2 + 1) * op3 + op1 + 1) * 3 + c] * 18
          + sharedExt[(( op2 - 1) * op3 + op1) * 3 + c] * -9 + 9 * sharedExt[( op2* op3 + op1) * 3 + c]
          + sharedExt[(( op2 + 1) * op3 + op1) * 3 + c] * 9;
          out[(j * w + i) * 3 + c] = (op4 / 9);

        }

    }
  }
}


int main()
{
  cv::Mat img_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = img_in.data;
  auto rows = img_in.rows;
  auto cols = img_in.cols;
  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat img_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;
  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );
  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 dim1( 32, 32 );
  dim3 dim2( 3 * (( cols - 1) / (dim1.x-2) + 1) , ( rows - 1 ) / (dim1.y-2) + 1 );
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );
  embossShared<<< dim2, dim1, 3*dim1.x*dim1.y >>>( rgb_d, out, cols, rows );
  hipMemcpy(g.data(), out, 3 * rows * cols, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  auto hipError_t = hipGetLastError();
  if (hipError_t != hipSuccess){
    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }
  else {
    std::cout << "No Errors!" << std::endl;
  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );
  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "Total: " << duration << "ms\n";
  cv::imwrite( "EmbossSharedOutput.jpg", img_out );
  hipFree( rgb_d);
  hipFree ( out);
  return 0;
}
