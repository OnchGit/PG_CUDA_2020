#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscaleShared ( unsigned char * in,   unsigned char * out, std::size_t w, std::size_t h) {

  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  auto op1 = threadIdx.x;
  auto op2 = threadIdx.y;

  extern __shared__ unsigned char sharedExt[];

  if( i < w && j < h ) {
		sharedExt[ (op2 * blockDim.x + op1) ] = (
		  307 * in[ 3 * ( j * w + i ) ]
		  + 604 * in[ 3 * ( j * w + i ) + 1 ]
		  + 113 * in[  3 * ( j * w + i ) + 2 ]
		) / 1024;
		__syncthreads();
		out[(j * w + i)] = sharedExt[(op2 * blockDim.x + op1)];
	}
}

int main()
{
  cv::Mat img_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = img_in.data;
  auto rows = img_in.rows;
  auto cols = img_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat img_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;
  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, rows * cols );
  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 dim1( 32, 32 );
  dim3 dim2(  (( cols - 1) / (dim1.x-2) + 1) , ( rows - 1 ) / (dim1.y-2) + 1 );
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );
  grayscaleShared<<< dim2, dim1, dim1.x*dim1.y >>>( rgb_d, out, cols, rows );
  hipMemcpy(g.data(), out, rows * cols, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  auto hipError_t = hipGetLastError();
  if (hipError_t != hipSuccess){
    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }
  else {
    std::cout << "No Errors!" << std::endl;
  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );
  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "Total: " << duration << "ms\n";
  cv::imwrite( "GrayscaleSharedOutput.jpg", img_out );
  hipFree( rgb_d);
  hipFree ( out);
  return 0;
}
