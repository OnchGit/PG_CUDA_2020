#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto op1 = blockIdx.x * blockDim.x + threadIdx.x;
  auto op2 = blockIdx.y * blockDim.y + threadIdx.y;
  if( op1 < cols && op2 < rows ) {
    g[ op2 * cols + op1 ] = (
      307 * rgb[ 3 * ( op2 * cols + op1 ) ]
      + 604 * rgb[ 3 * ( op2 * cols + op1 ) + 1 ]
      + 113 * rgb[  3 * ( op2 * cols + op1 ) + 2 ]
    ) / 1024;
  }
}

int main()
{
  cv::Mat img_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = img_in.data;
  auto rows = img_in.rows;
  auto cols = img_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat img_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;
  std::size_t size = img_in.cols * img_in.rows;
  std::size_t sizeRGB = 3 * img_in.cols * img_in.rows;
  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, rows * cols );
  hipStream_t streams[ 2 ];
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );
  hipMemcpyAsync( rgb_d, rgb, sizeRGB/2, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+sizeRGB/2, rgb+sizeRGB/2, sizeRGB/2, hipMemcpyHostToDevice, streams[ 1 ] );
  dim3 dim1( 32, 32 );
  dim3 dim2( (( cols ) / ((dim1.x - 2) + 1) ), (( rows ) / ((dim1.y - 2) + 1) ));
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );
  grayscale<<< dim2, dim1, 0, streams[ 0 ] >>>( rgb_d, out, cols, rows);
  grayscale<<< dim2, dim1, 0, streams[ 1 ] >>>( rgb_d+sizeRGB/2, out+size/2, cols, rows);
  hipMemcpyAsync( g.data(), out, size/2, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/2, hipMemcpyDeviceToHost, streams[ 1 ] );
  hipDeviceSynchronize();
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);
  auto hipError_t = hipGetLastError();
  if (hipError_t != hipSuccess){
    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }
  else {
    std::cout << "No Errors!" << std::endl;
  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );
  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "Total: " << duration << "ms\n";
  cv::imwrite( "GrayscaleStreamOutput.jpg", img_out );
  hipFree( rgb_d);
  hipFree ( out);
  return 0;
}
