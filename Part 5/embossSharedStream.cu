#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void embossShared ( unsigned char * data,   unsigned char * out, std::size_t w, std::size_t h) {
  auto op1 = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto op2 = blockIdx.y * (blockDim.y-2) + threadIdx.y;
  auto op3 = threadIdx.x;
  auto op4 = threadIdx.y;
  extern __shared__ unsigned char sharedExt[];
  if( op1 < w && op2 < h ) {
    sharedExt[3 * (op4 * blockDim.x + op3) ] = data[ 3 * ( op2 * w + op1 ) ];
    sharedExt[3 * (op4 * blockDim.x + op3) + 1 ] = data[ 3 * ( op2 * w + op1 ) + 1];
    sharedExt[3 * (op4 * blockDim.x + op3) + 2 ] = data[ 3 * ( op2 * w + op1 ) + 2 ];
    __syncthreads();
    auto op5 = blockDim.x;
    if( op3 > 0 && op3 < (blockDim.x - 1) && op4 > 0 && op4 < (blockDim.y - 1) )
    {
      for (auto c = 0; c < 3; ++c){
			auto op6 = sharedExt[((op4 - 1) * op5 + op3 - 1) * 3 + c] * -18 + sharedExt[((op4 - 1) * op5 + op3 + 1) * 3 + c] * 0
			+ sharedExt[( op4* op5 + op3 - 1) * 3 + c] * -9 + sharedExt[( op4 * op5 + op3 + 1) * 3 + c] * 9
			+ sharedExt[((op4 + 1) * op5 + op3 - 1) * 3 + c] * 0  + sharedExt[((op4 + 1) * op5 + op3 + 1) * 3 + c] * 18
			+ sharedExt[(( op4 - 1) * op5 + op3) * 3 + c] * -9 + 9 * sharedExt[( op4 * op5 + op3) * 3 + c]
			+ sharedExt[(( op4 + 1) * op5 + op3) * 3 + c] * 9;
			out[(op2 * w + op1) * 3 + c] = (op6 / 9);
		}
    }
  }
}

int main()
{
  cv::Mat img_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = img_in.data;
  auto rows = img_in.rows;
  auto cols = img_in.cols;
  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat img_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;
  std::size_t size = 3 * img_in.cols * img_in.rows;
  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );
  hipStream_t streams[ 2 ];
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );
  hipMemcpyAsync( rgb_d, rgb, size/2, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+size/2, rgb+size/2, size/2, hipMemcpyHostToDevice, streams[ 1 ] );
  dim3 dim1( 32, 32 );
  dim3 dim2( 3 * (( cols ) / ((dim1.x - 2) + 1) ), (( rows ) / ((dim1.y - 2) + 1) ));
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );
  embossShared<<< dim2, dim1, 3 * dim1.x * dim1.y, streams[ 0 ] >>>( rgb_d, out, cols, rows / 2 + 2);
  embossShared<<< dim2, dim1, 3 * dim1.x * dim1.y, streams[ 1 ] >>>( rgb_d+size/2, out+size/2, cols, rows / 2);
  hipMemcpyAsync( g.data(), out, size/2, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/2, hipMemcpyDeviceToHost, streams[ 1 ] );
  hipDeviceSynchronize();
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);
  auto hipError_t = hipGetLastError();
  if (hipError_t != hipSuccess){
    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }
  else {
    std::cout << "No Errors!" << std::endl;
  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );
  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "Total: " << duration << "ms\n";
  cv::imwrite( "EmbossSharedStreamOutput.jpg", img_out );
  hipFree( rgb_d);
  hipFree ( out);


  return 0;
}
